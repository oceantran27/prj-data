#include "hip/hip_runtime.h"
// Problem Link: https://leetcode.com/problems/longest-valid-parentheses/

// Refactored Solution

#include <string>
#include <vector>
#include <algorithm>
#include <numeric>
#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

auto solve(std::string const& problem) -> int 
{
    if (problem.empty()) return 0;
  
    auto const n       = problem.size();
    auto const op      = [](char c) { return c == '(' ? 1 : -1; }; 
    auto const it      = thrust::make_transform_iterator(problem.begin(), op);
    auto const mapping = thrust::device_vector<int>(it, it + n);
    auto const iota    = thrust::make_counting_iterator(0);
    auto const starts  = std::vector(iota, iota + n - 1);
  
    return std::accumulate(
        starts.cbegin(), starts.cend(), 0,
        [&mapping, iota, n](int max_so_far, int i) 
        {
            auto prefix = thrust::device_vector<int>(n - i);
            thrust::inclusive_scan(mapping.begin()+i, mapping.end(), prefix.begin());
            auto const indices = thrust::device_vector<int>(iota, iota + n - i);
  
            auto const max_for_prefix = std::transform_reduce(
                prefix.cbegin(), prefix.cend(), indices.cbegin(), 0, 
                thrust::maximum{},
                [] (auto p, auto i) { return p == 0 ? i : 0; });
  
            return std::max(max_so_far, max_for_prefix + 1);
        });
}

// Refactored Ideal Solution
// Note: thrust::make_scan_iterator and thrust::reduce_if do NOT exist

auto solve_ideal(std::string const& problem) -> int 
{
    if (problem.empty()) return 0;
  
    auto const n       = problem.size();
    auto const op      = [](char c) { return c == '(' ? 1 : -1; }; 
    auto const it      = thrust::make_transform_iterator(problem.begin(), op);
    auto const mapping = thrust::device_vector<int>(it, it + n);
    auto const iota    = thrust::make_counting_iterator(0);
    auto const starts  = std::vector(iota, iota + n - 1);
  
    return std::accumulate(
        starts.cbegin(), starts.cend(), 0,
        [&mapping, iota, n](int max_so_far, int i) 
        {
            auto const scan    = thrust::make_scan_iterator(mapping.begin() + i);
            auto const prefix  = thrust::device_vector<int>(scan, scan + n - i);
            auto const indices = thrust::device_vector<int>(iota, iota + n - i);
  
            auto const max_for_prefix = thrust::reduce_if(
                indices.cbegin(), indices.cend(), prefix.cbegin(), 0, 
                thrust::maximum{},       // reduction binary operation
                thrust::logical_not{});  // stencil unary predicate **
  
            return std::max(max_so_far, max_for_prefix + 1);
        });
}
